#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

#define N (1 << 17)  // 131072 элементов > 100000


// Последовательный Merge Sort

void merge(float *array, int left, int mid, int right, float *temp) {
    int i = left, j = mid, k = left;
    while (i < mid && j < right) {
        if (array[i] < array[j])
            temp[k++] = array[i++];
        else
            temp[k++] = array[j++];
    }
    while (i < mid) temp[k++] = array[i++];
    while (j < right) temp[k++] = array[j++];
    for (i = left; i < right; ++i)
        array[i] = temp[i];
}

void merge_sort_seq(float *array, int left, int right, float *temp) {
    if (right - left <= 1) return;
    int mid = (left + right) / 2;
    merge_sort_seq(array, left, mid, temp);
    merge_sort_seq(array, mid, right, temp);
    merge(array, left, mid, right, temp);
}


// Параллельный Bitonic Sort на CUDA

__global__ void bitonic_sort_step(float *dev_values, int j, int k) {
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    if (ixj > i) {
        if ((i & k) == 0) {
            if (dev_values[i] > dev_values[ixj]) {
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        } else {
            if (dev_values[i] < dev_values[ixj]) {
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

void bitonic_sort(float *values) {
    float *dev_values;
    size_t size = N * sizeof(float);
    hipMalloc((void**)&dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    dim3 blocks(N / 1024);
    dim3 threads(1024);

    for (int k = 2; k <= N; k <<= 1) {
        for (int j = k >> 1; j > 0; j >>= 1) {
            bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
            hipDeviceSynchronize();
        }
    }

    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}

int is_sorted(float *arr, int size) {
    for (int i = 0; i < size - 1; ++i)
        if (arr[i] > arr[i + 1]) return 0;
    return 1;
}

int compare_arrays(float *a, float *b, int size) {
    for (int i = 0; i < size; ++i)
        if (fabs(a[i] - b[i]) > 1e-5)
            return 0;
    return 1;
}


int main() {
    float *arr_seq = (float*)malloc(sizeof(float) * N);
    float *arr_par = (float*)malloc(sizeof(float) * N);
    float *temp = (float*)malloc(sizeof(float) * N);

    srand(time(NULL));
    for (int i = 0; i < N; ++i) {
        float val = (float)(rand() % 100000) / 100.0f;
        arr_seq[i] = val;
        arr_par[i] = val;
    }

    printf("Сортировка %d элементов...\n", N);

    // Последовательная сортировка
    clock_t start_seq = clock();
    merge_sort_seq(arr_seq, 0, N, temp);
    clock_t end_seq = clock();
    printf("Sequential time: %.4f\n", (double)(end_seq - start_seq) / CLOCKS_PER_SEC);

    // Параллельная сортировка
    clock_t start_par = clock();
    bitonic_sort(arr_par);
    clock_t end_par = clock();
    printf("Parallel time: %.4f\n", (double)(end_par - start_par) / CLOCKS_PER_SEC);

    // Проверка
    if (!is_sorted(arr_seq, N)) printf("❌ Последовательная сортировка НЕ верна\n");
    else printf("✅ Последовательная сортировка верна\n");

    if (!is_sorted(arr_par, N)) printf("❌ Параллельная сортировка НЕ верна\n");
    else printf("✅ Параллельная сортировка верна\n");

    if (compare_arrays(arr_seq, arr_par, N))
        printf("✅ Результаты сортировок совпадают\n");
    else
        printf("❌ Результаты сортировок не совпадают\n");

    free(arr_seq);
    free(arr_par);
    free(temp);
    return 0;
}
